#include <iostream>
#include <hip/hip_runtime.h>



__global__ void hellocud() {
    //fprintf(stderr, "cudaDeviceReset failed!");
    //std::cout << "Hello CUDA! FUNCCC!" << std::endl;
    printf("Hello CUDA Func!\n");
    //cudaPrintfDisplay("cudaPrintfDisplay");
}

void launchKernel() {
    hellocud<<<2, 2>>>();
    hipDeviceSynchronize(); // Çekirdek fonksiyonun tamamlanmasını bekle
}

int main() {
    
    launchKernel();

    hipError_t res = hipInit(0);
    std::cout << "Hello CUDA! Result: " << res << std::endl;


    return 0;
}

/*
Ekran çıktısı:
Hello CUDA! Result: 100
*/
